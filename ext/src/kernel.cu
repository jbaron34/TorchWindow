#include "hip/hip_runtime.h"
#include "twkernel.h"
#include <stdio.h>

#define BLOCK_SIZE 8
#define BLOCK_WIDTH BLOCK_SIZE
#define BLOCK_HEIGHT BLOCK_SIZE
#define NUM_OUT_CHANNELS 4

__global__
void dev_copy( const float  *in,
               float        *out,
               int           inDimA,
               int           inDimB,
               int           inDimC,
               int           outDimA,
               int           outDimB)
{
	int x, y, z,
        outIdx;
    float outValue;

	x = threadIdx.x + BLOCK_SIZE * blockIdx.x;
	y = threadIdx.y + BLOCK_SIZE * blockIdx.y;
	z = threadIdx.z;

    if( x >= inDimA  || y >= inDimB ||
        x >= outDimA || y >= outDimB
    ){
        return;
    }

    if (z >= inDimC ){
        outValue = 1.0f;
    }else{
        int inIdx;
        inIdx = x * inDimB * inDimC + y * inDimC + z;
        outValue = in[inIdx];
    }

    outIdx = x * outDimB * NUM_OUT_CHANNELS + y * NUM_OUT_CHANNELS + z;
    out[outIdx] = outValue;
}

void transposeCopy(
    const float *input,
    float *output,
    int inDimA,
    int inDimB,
    int inDimC,
    int outDimA,
    int outDimB,
    TransposeCopyType type
){
    switch(type){
        case transposeCopyXYZ:
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, NUM_OUT_CHANNELS);
        dim3 dimGrid(
            min(inDimA, outDimA) / BLOCK_SIZE + 1,
            min(inDimB, outDimB) / BLOCK_SIZE + 1
        );

        dev_copy<<<dimGrid, dimBlock>>>(
            input, output,
            inDimA, inDimB, inDimC,
            outDimA, outDimB
        );
    }

}